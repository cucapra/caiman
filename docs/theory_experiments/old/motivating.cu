
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(int* array, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	array[tid] = array[tid] * array[tid];
}

__host__ void caller()
{
	int* array = nullptr;
	hipMalloc(& array, sizeof(int));
	int element = 5;
	printf("element: %i\n", element);
	hipMemcpy(array, & element, sizeof(int), hipMemcpyHostToDevice);
	square<<<1, 1>>>(array, 1);
	hipMemcpy(& element, array, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(array);
	printf("element: %i\n", element);
}

__device__ __host__ void helper()
{
	
}

//https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
//https://leimao.github.io/blog/CUDA-Stream/
//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
//https://stackoverflow.com/questions/21986542/is-cudamallocmanaged-slower-than-cudamalloc
//https://stackoverflow.com/questions/39782746/why-is-nvidia-pascal-gpus-slow-on-running-cuda-kernels-when-using-cudamallocmana/40011988#40011988
//https://stackoverflow.com/questions/65501537/cudamallocmanaged-unified-memory-with-cublas
//https://developer.nvidia.com/blog/how-overlap-data-transfers-cuda-cc/
//https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
//https://stackoverflow.com/questions/8473617/are-cuda-kernel-calls-synchronous-or-asynchronous
__global__ void task_a(int* data, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	data[tid] += 1;
}

__host__ void task_b(int* data, int n)
{
	for (int tid = 0; tid < n; tid++)
		data[tid] += 1;
}

__host__ void task_c(int* data_a, const int* data_b, int n)
{
	for (int tid = 0; tid < n; tid++)
		data_a[tid] += data_b[tid];
}


__global__ void task_b_device(int* data, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	data[tid] += 1;
}

__global__ void task_c_device(int* data_a, const int* data_b, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	data_a[tid] += data_b[tid];
}

__global__ void task_d(int* data, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= n)
		return;
	data[tid] += 1;
}

__host__ void schedule_a(int input, int* output)
{
	int* cpu_data_a = (int*) malloc(sizeof(int));
	int* cpu_data_b = (int*) malloc(sizeof(int));
	int* gpu_data_a = NULL;
	//int* gpu_data_b = NULL;
	hipMalloc(& gpu_data_a, sizeof(int));
	//cudaMalloc(& gpu_data_b, sizeof(int));

	memcpy(cpu_data_a, & input, sizeof(int));
	memcpy(cpu_data_b, & input, sizeof(int));
	hipMemcpy(gpu_data_a, & input, sizeof(int), hipMemcpyHostToDevice);
	//cudaMemcpy(gpu_data_b, & input, sizeof(int), cudaMemcpyHostToDevice);

	printf("Line %i: %i\n", __LINE__, input);
	printf("Line %i: %i\n", __LINE__, * cpu_data_a);
	printf("Line %i: %i\n", __LINE__, * cpu_data_b);

	task_a<<<1, 1>>>(gpu_data_a, 1);
	task_b(cpu_data_b, 1);
	hipMemcpy(cpu_data_a, gpu_data_a, sizeof(int), hipMemcpyDeviceToHost);
	printf("Line %i: %i\n", __LINE__, * cpu_data_a);
	printf("Line %i: %i\n", __LINE__, * cpu_data_b);
	task_c(cpu_data_a, cpu_data_b, 1);
	printf("Line %i: %i\n", __LINE__, * cpu_data_a);
	printf("Line %i: %i\n", __LINE__, * cpu_data_b);
	hipMemcpy(gpu_data_a, cpu_data_a, sizeof(int), hipMemcpyHostToDevice);
	task_d<<<1, 1>>>(gpu_data_a, 1);

	hipMemcpy(output, gpu_data_a, sizeof(int), hipMemcpyDeviceToHost);

	free(cpu_data_a);
	free(cpu_data_b);
	hipFree(gpu_data_a);
	//cudaFree(gpu_data_b);
}

__host__ void schedule_a2(int input, int* output)
{
	int* data_a = (int*) malloc(sizeof(int));
	int* data_b = (int*) malloc(sizeof(int));
	hipMallocManaged(& data_a, sizeof(int));
	hipMallocManaged(& data_b, sizeof(int));

	memcpy(data_a, & input, sizeof(int));
	memcpy(data_b, & input, sizeof(int));

	printf("Line %i: %i\n", __LINE__, input);
	printf("Line %i: %i\n", __LINE__, * data_a);
	printf("Line %i: %i\n", __LINE__, * data_b);

	task_a<<<1, 1>>>(data_a, 1);
	task_b(data_b, 1);
	hipStreamSynchronize(0);
	printf("Line %i: %i\n", __LINE__, * data_a);
	printf("Line %i: %i\n", __LINE__, * data_b);
	task_c(data_a, data_b, 1);
	printf("Line %i: %i\n", __LINE__, * data_a);
	printf("Line %i: %i\n", __LINE__, * data_b);
	task_d<<<1, 1>>>(data_a, 1);
	hipStreamSynchronize(0);

	memcpy(output, data_a, sizeof(int));

	hipFree(data_a);
	hipFree(data_b);
}

__host__ void schedule_b(int input, int* output)
{
	int* cpu_data_a = (int*) malloc(sizeof(int));
	int* cpu_data_b = (int*) malloc(sizeof(int));
	int* gpu_data_a = NULL;
	//int* gpu_data_b = NULL;
	hipMalloc(& gpu_data_a, sizeof(int));
	//cudaMalloc(& gpu_data_b, sizeof(int));

	memcpy(cpu_data_a, & input, sizeof(int));
	memcpy(cpu_data_b, & input, sizeof(int));
	hipMemcpyAsync(gpu_data_a, & input, sizeof(int), hipMemcpyHostToDevice, 0);
	//cudaMemcpy(gpu_data_b, & input, sizeof(int), cudaMemcpyHostToDevice);

	task_a<<<1, 1, 0, 0>>>(gpu_data_a, 1);
	hipMemcpyAsync(cpu_data_a, gpu_data_a, sizeof(int), hipMemcpyDeviceToHost, 0);
	task_b(cpu_data_b, 1);
	hipStreamSynchronize(0);
	task_c(cpu_data_a, cpu_data_b, 1);
	hipMemcpyAsync(gpu_data_a, cpu_data_a, sizeof(int), hipMemcpyHostToDevice, 0);
	task_d<<<1, 1>>>(gpu_data_a, 1);

	hipMemcpy(output, gpu_data_a, sizeof(int), hipMemcpyDeviceToHost);

	free(cpu_data_a);
	free(cpu_data_b);
	hipFree(gpu_data_a);
	//cudaFree(gpu_data_b);
}

__host__ void schedule_c(int input, int* output)
{
	int* gpu_data_a = NULL;
	int* gpu_data_b = NULL;
	hipMalloc(& gpu_data_a, sizeof(int));
	hipMalloc(& gpu_data_b, sizeof(int));

	hipMemcpyAsync(gpu_data_a, & input, sizeof(int), hipMemcpyHostToDevice, 0);
	hipMemcpyAsync(gpu_data_b, & input, sizeof(int), hipMemcpyHostToDevice, 0);

	task_a<<<1, 1, 0, 0>>>(gpu_data_a, 1);
	task_b_device<<<1, 1, 0, 0>>>(gpu_data_b, 1);
	task_c_device<<<1, 1, 0, 0>>>(gpu_data_a, gpu_data_b, 1);
	task_d<<<1, 1, 0, 0>>>(gpu_data_a, 1);

	hipMemcpy(output, gpu_data_a, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(gpu_data_a);
	hipFree(gpu_data_b);
}

int main()
{
	caller();
	int input = 3, output = 0;
	schedule_a(input, & output);
	printf("schedule_a: %i in %i out\n", input, output);
	output = 0;
	schedule_a2(input, & output);
	printf("schedule_a2: %i in %i out\n", input, output);
	output = 0;
	schedule_b(input, & output);
	printf("schedule_b: %i in %i out\n", input, output);
	output = 0;
	schedule_c(input, & output);
	printf("schedule_c: %i in %i out\n", input, output);
	return 0;
}